#include "hip/hip_runtime.h"
#include <immintrin.h>
#include <omp.h>
#include <pthread.h>

#include <cassert>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <queue>
#include <thread>
#include <utility>
#include <vector>

#ifdef DEBUG
#define DEBUG_PRINT(fmt, args...) fprintf(stderr, fmt, ##args);
#define DEBUG_MSG(str) std::cout << str << "\n";
#else
#define DEBUG_PRINT(fmt, args...)
#define DEBUG_MSG(str)
#endif  // DEBUG

#ifdef TIMING
#include <ctime>
#define TIMING_START(arg)          \
    struct timespec __start_##arg; \
    clock_gettime(CLOCK_MONOTONIC, &__start_##arg);
#define TIMING_END(arg)                                                                       \
    {                                                                                         \
        struct timespec __temp_##arg, __end_##arg;                                            \
        double __duration_##arg;                                                              \
        clock_gettime(CLOCK_MONOTONIC, &__end_##arg);                                         \
        if ((__end_##arg.tv_nsec - __start_##arg.tv_nsec) < 0) {                              \
            __temp_##arg.tv_sec = __end_##arg.tv_sec - __start_##arg.tv_sec - 1;              \
            __temp_##arg.tv_nsec = 1000000000 + __end_##arg.tv_nsec - __start_##arg.tv_nsec;  \
        } else {                                                                              \
            __temp_##arg.tv_sec = __end_##arg.tv_sec - __start_##arg.tv_sec;                  \
            __temp_##arg.tv_nsec = __end_##arg.tv_nsec - __start_##arg.tv_nsec;               \
        }                                                                                     \
        __duration_##arg = __temp_##arg.tv_sec + (double)__temp_##arg.tv_nsec / 1000000000.0; \
        printf("%s took %lfs.\n", #arg, __duration_##arg);                                    \
    }
#else
#define TIMING_START(arg)
#define TIMING_END(arg)
#endif  // TIMING

#define block_size 64
#define half_bs 32
#define div_block 2
const int INF = ((1 << 30) - 1);

struct edge_t {
    int src;
    int dst;
    int w;
};

int blk_idx(int r, int c, int nblocks);

void proc(int *blk_dist, int s_i, int e_i, int s_j, int e_j, int k, int nblocks, int ncpus);

__global__ void proc_1_glob(int *blk_dist, int k, int nblocks);
__global__ void proc_2_glob(int *blk_dist, int s, int k, int nblocks);
__global__ void proc_3_glob(int *blk_dist, int s_i, int s_j, int k, int nblocks);

int main(int argc, char **argv) {
    assert(argc == 3);

    char *input_filename = argv[1];
    char *output_filename = argv[2];
    FILE *input_file;
    FILE *output_file;
    int ncpus = omp_get_max_threads();
    int V, E;
    edge_t *edge;
    int *dist;
    int VP;
    int nblocks;
    int *blk_dist;
    int *blk_dist_dev;

    TIMING_START(hw3_1);

    /* input */
    TIMING_START(input);
    input_file = fopen(input_filename, "rb");
    assert(input_file);
    fread(&V, sizeof(int), 1, input_file);
    fread(&E, sizeof(int), 1, input_file);
    edge = (edge_t *)malloc(sizeof(edge_t) * E);
    fread(edge, sizeof(edge_t), E, input_file);
    dist = (int *)malloc(sizeof(int) * V * V);
    fclose(input_file);
    DEBUG_PRINT("vertices: %d\nedges: %d\n", V, E);
    TIMING_END(input);

    /* calculate */
    TIMING_START(calculate);
    nblocks = (int)ceilf(float(V) / block_size);
    VP = nblocks * block_size;
    blk_dist = (int *)malloc(sizeof(int) * VP * VP);

    for (int i = 0; i < VP; i++) {
        for (int j = 0; j < VP; j++) {
            if (i == j)
                blk_dist[blk_idx(i, j, nblocks)] = 0;
            else
                blk_dist[blk_idx(i, j, nblocks)] = INF;
        }
    }

    for (int i = 0; i < E; i++) {
        blk_dist[blk_idx(edge[i].src, edge[i].dst, nblocks)] = edge[i].w;
    }

    hipHostRegister(blk_dist, sizeof(int) * VP * VP, hipHostRegisterDefault);
    hipMalloc(&blk_dist_dev, sizeof(int) * VP * VP);
    hipMemcpy(blk_dist_dev, blk_dist, sizeof(int) * VP * VP, hipMemcpyHostToDevice);

    dim3 blk(block_size / div_block, block_size / div_block);
    for (int k = 0, nk = nblocks - 1; k < nblocks; k++, nk--) {
        /* Phase 1 */
        proc_1_glob<<<1, blk>>>(blk_dist_dev, k, nblocks);
        /* Phase 2 */
        proc_2_glob<<<nblocks, blk>>>(blk_dist_dev, 0, k, nblocks);
        /* Phase 3 */
        proc_3_glob<<<dim3(nblocks, nblocks), blk>>>(blk_dist_dev, 0, 0, k, nblocks);
    }

    hipMemcpy(blk_dist, blk_dist_dev, sizeof(int) * VP * VP, hipMemcpyDeviceToHost);

    /* Copy result to dist */
    for (int i = 0; i < V; i++) {
        for (int j = 0; j < V; j++) {
            dist[i * V + j] = min(blk_dist[blk_idx(i, j, nblocks)], INF);
        }
    }

    TIMING_END(calculate);

    /* output */
    TIMING_START(output);
    output_file = fopen(output_filename, "w");
    assert(output_file);
    fwrite(dist, sizeof(int), V * V, output_file);
    fclose(output_file);
    TIMING_END(output);
    TIMING_END(hw3_1);

    /* finalize */
    free(edge);
    free(dist);
    free(blk_dist);
    hipFree(blk_dist_dev);
    return 0;
}

int blk_idx(int r, int c, int nblocks) {
    return ((r / block_size) * nblocks + (c / block_size)) * block_size * block_size + (r % block_size) * block_size + (c % block_size);
}

void proc(int *blk_dist, int s_i, int e_i, int s_j, int e_j, int k, int nblocks, int ncpus) {
#pragma omp parallel for num_threads(ncpus) schedule(static) default(shared) collapse(2)
    for (int i = s_i; i < e_i; i++) {
        for (int j = s_j; j < e_j; j++) {
            int *ik_ptr = blk_dist + (i * nblocks + k) * block_size * block_size;
            int *ij_ptr = blk_dist + (i * nblocks + j) * block_size * block_size;
            int *kj_ptr = blk_dist + (k * nblocks + j) * block_size * block_size;
            for (int b = 0; b < block_size; b++) {
                for (int r = 0; r < block_size; r++) {
#pragma omp simd
                    for (int c = 0; c < block_size; c++) {
                        ij_ptr[r * block_size + c] = std::min(ij_ptr[r * block_size + c], ik_ptr[r * block_size + b] + kj_ptr[b * block_size + c]);
                    }
                }
            }
        }
    }
}

__global__ void proc_1_glob(int *blk_dist, int k, int nblocks) {
    __shared__ int k_k_sm[block_size][block_size];

    int r = threadIdx.y;
    int c = threadIdx.x;
    int *k_k_ptr = blk_dist + (k * nblocks + k) * (block_size * block_size);
    int tmp;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_k_sm[r + rr * half_bs][c + cc * half_bs] = k_k_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs];
        }
    }
    __syncthreads();

    // #pragma unroll
    for (int b = 0; b < block_size; b++) {
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                tmp = k_k_sm[r + rr * half_bs][b] + k_k_sm[b][c + cc * half_bs];
                if (tmp < k_k_sm[r + rr * half_bs][c + cc * half_bs])
                    k_k_sm[r + rr * half_bs][c + cc * half_bs] = tmp;
            }
        }
        __syncthreads();
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_k_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs] = k_k_sm[r + rr * half_bs][c + cc * half_bs];
        }
    }
}
__global__ void proc_2_glob(int *blk_dist, int s, int k, int nblocks) {
    __shared__ int i_k_sm[block_size][block_size];
    __shared__ int k_j_sm[block_size][block_size];
    __shared__ int k_k_sm[block_size][block_size];

    int i = s + blockIdx.x;
    int j = s + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int *i_k_ptr = blk_dist + (i * nblocks + k) * (block_size * block_size);
    int *k_j_ptr = blk_dist + (k * nblocks + j) * (block_size * block_size);
    int *k_k_ptr = blk_dist + (k * nblocks + k) * (block_size * block_size);
    int tmp_i_k, tmp_k_j;

    if (i == k)
        return;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            i_k_sm[r + rr * half_bs][c + cc * half_bs] = i_k_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs];
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_j_sm[r + rr * half_bs][c + cc * half_bs] = k_j_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs];
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_k_sm[r + rr * half_bs][c + cc * half_bs] = k_k_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs];
        }
    }
    __syncthreads();

    // #pragma unroll
    for (int b = 0; b < block_size; b++) {
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                tmp_i_k = i_k_sm[r + rr * half_bs][b] + k_k_sm[b][c + cc * half_bs];
                if (tmp_i_k < i_k_sm[r + rr * half_bs][c + cc * half_bs])
                    i_k_sm[r + rr * half_bs][c + cc * half_bs] = tmp_i_k;
                tmp_k_j = k_k_sm[r + rr * half_bs][b] + k_j_sm[b][c + cc * half_bs];
                if (tmp_k_j < k_j_sm[r + rr * half_bs][c + cc * half_bs])
                    k_j_sm[r + rr * half_bs][c + cc * half_bs] = tmp_k_j;
            }
        }
        __syncthreads();
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            i_k_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs] = i_k_sm[r + rr * half_bs][c + cc * half_bs];
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_j_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs] = k_j_sm[r + rr * half_bs][c + cc * half_bs];
        }
    }
}
__global__ void proc_3_glob(int *blk_dist, int s_i, int s_j, int k, int nblocks) {
    __shared__ int i_k_sm[block_size][block_size];
    __shared__ int k_j_sm[block_size][block_size];

    int i = s_i + blockIdx.y;
    int j = s_j + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int *i_k_ptr = blk_dist + (i * nblocks + k) * (block_size * block_size);
    int *i_j_ptr = blk_dist + (i * nblocks + j) * (block_size * block_size);
    int *k_j_ptr = blk_dist + (k * nblocks + j) * (block_size * block_size);
    int loc[div_block][div_block], tmp;

    if (i == k || j == k)
        return;

#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            i_k_sm[r + rr * half_bs][c + cc * half_bs] = i_k_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs];
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            k_j_sm[r + rr * half_bs][c + cc * half_bs] = k_j_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs];
        }
    }
    __syncthreads();
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            loc[rr][cc] = i_j_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs];
        }
    }

    // #pragma unroll
    for (int b = 0; b < block_size; b++) {
#pragma unroll
        for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
            for (int cc = 0; cc < div_block; cc++) {
                tmp = i_k_sm[r + rr * half_bs][b] + k_j_sm[b][c + cc * half_bs];
                if (tmp < loc[rr][cc])
                    loc[rr][cc] = tmp;
            }
        }
    }
#pragma unroll
    for (int rr = 0; rr < div_block; rr++) {
#pragma unroll
        for (int cc = 0; cc < div_block; cc++) {
            i_j_ptr[(r + rr * half_bs) * block_size + c + cc * half_bs] = loc[rr][cc];
        }
    }
}